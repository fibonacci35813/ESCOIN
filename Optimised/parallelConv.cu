#include "hip/hip_runtime.h"
%%cuda --name Convolution.cu

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define H 6 // dimension of the input features 
#define W 3
#define warp_size 8

int *value;
int *colidx;
int *rowptr;
int *rowidx;
__managed__ int position_value=0;
__managed__ int position_rowptr=1;
__managed__ int position_colidx=0;
__managed__ int position_rowidx=0;

 
void sparesify(int* matrix)
{
    value=(int*)malloc(sizeof(int)*(W*W));
    rowptr=(int*)malloc(sizeof(int)*(W+1));
    colidx=(int*)malloc(sizeof(int)*(W*W));
    rowidx=(int*)malloc(sizeof(int)*(W));
    int NNZ = 0;
    rowptr[0]=0;
    for (int i = 0; i < W; i++) {
        for (int j = 0; j < W; j++) {
            if (matrix[i*W+j] != 0) {
                value[position_value++] = matrix[i*W+j];
                colidx[position_colidx++] = j;
                rowidx[position_rowidx++] = i;
                NNZ++;
            }
        }
        rowptr[position_rowptr++] = NNZ;
    }
}

int *SPConv_serial(int input_features[H*H], int weight[], int window_size)
{
	int j1, j2;
	sparesify(weight);
   
	int *output_matrix = (int *)calloc(window_size*window_size, sizeof(int *)); 
    	
	for (int i=0;i<position_rowptr-1;i++) 
	{
    		int row = i;
    		for (int j=rowptr[i];j<rowptr[i+1];j++) 
    		{
      			int mat_value = value[j];
      			int col = colidx[j];
      			for (j1=0; j1<window_size;j1++) 
      			{
        			for (j2=0; j2<window_size;j2++) 
        			{
          				output_matrix[j1*window_size + j2] = output_matrix[j1*window_size + j2]  + input_features[(j1+row)*H + (j2+col)] * mat_value;
			        }
			       		
		        }
    		}
	}

	return output_matrix;

}

__global__ void conv_mul_parallel(int* d_if, int *val, int *row, int * col_id, int * row_id, int* d_of, int window_size)
{
    extern __shared__ int psum[] ;
    __shared__ int active_tid;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
    {
        for (int idx = 1; idx < position_rowptr; idx++)
            psum[idx] = 0;
    }

    if (tid <= position_rowptr - 1)
    {
        for (int k=0; k < position_value; k++)
        {
            int mat_value = val[k];
            int row = row_id[k];
            int col = col_id[k];

            // printf("\n mat-value: %d", mat_value);
            // printf("\n row num: %d", row);
            // printf("\n col num: %d\n", col);
            
            for (int j1 = 0; j1 < window_size; j1++)
            {
                for (int j2=0; j2 < window_size; j2++)
                {
                    if (tid == j2 % warp_size)
                    {
                        // printf("\nActive threadID: %d\n", j2);
                        active_tid = tid;
                        psum[tid] = d_if[(row + j1) * H + (col + j2)] * mat_value;
                        // printf("\npsum value: %d\n", psum[tid]);
                    }

                    __syncthreads();
                    d_of[j1 * window_size + active_tid] += psum[active_tid];

                }
                
            }
        }

    }

}

int* copy(int* arr, int size)
{
    int * ret = (int*)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++)
        ret[i] = arr[i];
    return ret;
    
}

int main(void) 
{
	int w[] = {0, 0, 0, 0, 0, 2, 3, 0, 0};
   
    /*int ifmaps[H * H] = {1,2,3,4,5,6,
            7,8,9,10,11,12,
            13,14,15,16,17,18,
            19,20,21,22,23,24,
            25};*/

    int ifmaps[H * H] = {1,2,3,4,5,6,
            7,8,9,10,11,12,
            13,14,15,16,17,18,
            19,20,21,22,23,24,
            25, 26, 27, 28, 29, 30, 31, 32, 33, 34, 35, 36 };
	int op_size = H - W + 1;
    sparesify(w);
	//int *ofmaps;
	//ofmaps = SPConv_serial(ifmaps, w, op_size);
    int* d_ifmaps, * d_of;
    int * d_ofmaps = (int*)malloc(op_size * op_size * sizeof(int));

    // printf("\nValue[]: ");
    // for(int i=0;i<position_value;i++) 
	// {
    // 	printf("%d ", value[i]);
  	// }

    // printf("\nRow[]: ");
    // for(int i=0;i<position_rowidx;i++) 
	// {
    // 	printf("%d ", rowidx[i]);
  	// }

    // printf("\nCol[]: ");
    // for(int i=0;i<position_colidx;i++) 
	// {
    // 	printf("%d ", colidx[i]);
  	// }

    // printf("\n\n");

    hipMalloc((void**)&d_ifmaps, H * H * sizeof(int));
    hipMalloc((void**)&d_of, op_size * op_size * sizeof(int));

    hipMemcpy(d_ifmaps, ifmaps, H * H * sizeof(int), hipMemcpyHostToDevice);

    int* h_value, * h_rowptr, * h_colidx, * h_rowidx;
    int* d_value, * d_rowptr, * d_colidx, * d_rowidx;
  
    h_value = copy(value, position_value);
    h_rowptr = copy(rowptr, position_rowptr);
    h_colidx = copy(colidx, position_colidx);
    h_rowidx = copy(rowidx, position_rowidx);

    hipMalloc((void**)&d_value, position_value * sizeof(int));
    hipMalloc((void**)&d_rowptr, position_rowptr * sizeof(int));
    hipMalloc((void**)&d_colidx, position_colidx * sizeof(int));
    hipMalloc((void**)&d_rowidx, position_rowidx * sizeof(int));

    hipMemcpy(d_value, h_value, position_value * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rowptr, h_rowptr, position_rowptr * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colidx, h_colidx, position_colidx * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rowidx, h_rowidx, position_rowidx * sizeof(int), hipMemcpyHostToDevice);

    int tpb = 8;
    int bpg = (tpb + position_rowptr - 1) / tpb;

    conv_mul_parallel << <bpg, tpb, position_rowptr *sizeof(int) >> > (d_ifmaps, d_value, d_rowptr, d_colidx, d_rowidx, d_of, op_size);

    hipMemcpy(d_ofmaps, d_of, op_size * op_size * sizeof(int), hipMemcpyDeviceToHost);
  
    printf("\nOutput maxtrix: \n");
	for(int i=0;i<op_size;i++) 
	{
    		for(int j=0;j<op_size;j++) 
    		{
      			printf("%d ", d_ofmaps[i* op_size + j]);
    		}
    		printf("\n");
  	}
	return 0;
}
